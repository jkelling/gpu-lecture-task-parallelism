#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "test_fixture.hpp"

#include "hip/hip_runtime_api.h"

#include <iostream>
#include <chrono>
#include <vector>

__global__ void increment_kernel(int *g_data, int inc_value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

double increment_by_one(std::int32_t* data,
		      std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);


  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);
  increment_kernel<<<blocks, threads>>>(d_a, value);
  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);

  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}

double parallel_increment_by_one(std::int32_t* data,
                             std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  std::vector<hipStream_t> streams(2);
  for( hipStream_t& el : streams ){
    hipStreamCreate(&el);
  }

  // allocate device memory
  int * h_a[2];
  for(int s = 0;s < streams.size();++s){
    checkCudaErrors(hipHostAlloc((void **)&h_a[s], nbytes/2, hipHostMallocPortable));
    std::copy(data+s*(size/2), data+(s+1)*(size/2),h_a[s]);
  }

  // allocate device memory
  int *d_a[2];
  for(int s = 0;s < streams.size();++s){
    checkCudaErrors(hipMalloc((void **)&d_a[s], nbytes));
    checkCudaErrors(hipMemset(d_a[s], 255, nbytes));
  }

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / (2*threads.x), 1);


  auto start = std::chrono::high_resolution_clock::now();

  for(int s = 0;s < streams.size();++s){
    hipMemcpyAsync(d_a[s], h_a[s], nbytes/2, hipMemcpyHostToDevice, streams[s]);
    increment_kernel<<<blocks, threads,0,streams[s]>>>(d_a[s], value);
    hipMemcpyAsync( h_a[s],d_a[s], nbytes/2, hipMemcpyDeviceToHost, streams[s]);
  }

  hipDeviceSynchronize();

  auto end = std::chrono::high_resolution_clock::now();

  for(int s = 0;s < streams.size();++s){
    checkCudaErrors(hipFree(d_a[s]));
    std::copy(h_a[s],h_a[s]+size/2,data+s*(size/2));
    checkCudaErrors(hipHostFree(h_a[s]));
  }

  return (end - start).count();
}


TEST_CASE_METHOD(array_fixture, "simple_cuda_increment_works" ) {

  increment_by_one(ints.data(), ints.size());
  
  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 1);
  
}

TEST_CASE_METHOD(array_fixture, "streams_increment_works" ) {

  parallel_increment_by_one(ints.data(), ints.size());

  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 1);

}

TEST_CASE_METHOD(array_fixture, "compare_times" ) {

  auto serial = increment_by_one(ints.data(), ints.size());
  auto parallel = parallel_increment_by_one(ints.data(), ints.size());

  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 2);

  REQUIRE(parallel < serial);
}
