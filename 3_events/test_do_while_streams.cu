#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "test_fixture.hpp"

#include "hip/hip_runtime_api.h"

#include <iostream>
#include <chrono>
#include <vector>

__global__ void increment_kernel(int *g_data, int inc_value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

double increment_by_one(std::int32_t* data,
		      std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);

  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);
  increment_kernel<<<blocks, threads>>>(d_a, value);
  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);

  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}

double parallel_increment_by_one(std::int32_t* data,
                             std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  std::vector<hipStream_t> streams(2);
  for( hipStream_t& el : streams ){
    hipStreamCreate(&el);
  }

  int * h_a[2];
  for(int s = 0;s < streams.size();++s){
    checkCudaErrors(hipHostAlloc((void **)&h_a[s], nbytes/2, hipHostMallocPortable));
    std::copy(data+s*(size/2), data+(s+1)*(size/2), h_a[s]);
  }

  // allocate device memory
  int *d_a[2];
  for(int s = 0;s < streams.size();++s){
    checkCudaErrors(hipMalloc((void **)&d_a[s], nbytes/2));
    checkCudaErrors(hipMemset(d_a[s], 255, nbytes/2));
  }

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / (2*threads.x), 1);

  hipEvent_t kstart, kend;
  hipEventCreate(&kstart);
  hipEventCreate(&kend);

  hipEventRecord(kstart, streams[0]);
  hipMemcpyAsync(d_a[0], h_a[0], nbytes/2, hipMemcpyHostToDevice, streams[0]);
  hipMemcpyAsync(d_a[1], h_a[1], nbytes/2, hipMemcpyHostToDevice, streams[1]);

  increment_kernel<<<blocks, threads,0,streams[0]>>>(d_a[0], value);
  increment_kernel<<<blocks, threads,0,streams[1]>>>(d_a[1], value);

  hipMemcpyAsync(h_a[1],d_a[1], nbytes/2, hipMemcpyDeviceToHost, streams[1]);
  hipMemcpyAsync(h_a[0],d_a[0], nbytes/2, hipMemcpyDeviceToHost, streams[0]);
  hipEventRecord(kend,streams[0]);

  int counter = 0;
  while(hipEventQuery(kend) == hipErrorNotReady){
    counter++;
  }
  
  hipDeviceSynchronize();


  float rvalue = 0.;
  hipEventElapsedTime(&rvalue,kstart,kend);

  for(int s = 0;s < streams.size();++s){
    std::copy(h_a[s], h_a[s]+size/2,data+s*(size/2));
    checkCudaErrors(hipHostFree(h_a[s]));
    checkCudaErrors(hipFree(d_a[s]));
  }

  return counter;
}


TEST_CASE_METHOD(array_fixture, "simple_cuda_increment_works" ) {

  increment_by_one(ints.data(), ints.size());
  
  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 1);
  
}

TEST_CASE_METHOD(array_fixture, "streams_increment_works" ) {

  parallel_increment_by_one(ints.data(), ints.size());

  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 1);

}

TEST_CASE_METHOD(array_fixture, "check_counter" ) {

  auto counter = parallel_increment_by_one(ints.data(), ints.size());

  REQUIRE(counter > 0);
  
  std::cout << "counter  : " << counter << "\n";
           
}
