#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "test_fixture.hpp"

#include "hip/hip_runtime_api.h"

#include <iostream>
#include <chrono>
#include <vector>

__global__ void increment_kernel(int *g_data, int inc_value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value + 10e6*(int)logf(sinf((float)idx) + tanf((float)idx));
}

double increment_by_one(std::int32_t* data,
		      std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);


  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);
  increment_kernel<<<blocks, threads>>>(d_a, value);
  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);

  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}

double parallel_increment_by_one(std::int32_t* data,
                             std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);

  std::vector<hipStream_t> streams(4);
  for( hipStream_t& el : streams ){
    hipStreamCreate(&el);
  }

  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);

  for(int s = 0;s < streams.size();++s){
    std::cout << "increment_kernel " << s << "/" << streams.size() << "\n";
    increment_kernel<<<blocks, threads,0,streams[s]>>>(d_a, value);
  }

  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);
  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}

std::size_t count_up(std::int32_t* data,
                             std::size_t size){

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  int *h_a=0;
  checkCudaErrors(hipHostAlloc((void **)&h_a, nbytes, hipHostMallocPortable));
  std::fill(h_a,h_a + size,42);
  std::copy(data,data+size,h_a);

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);
  unsigned long int counter=0;

  hipEvent_t kstart, kend;
  hipEventCreate (&kstart);
  hipEventCreate (&kend);

  hipEventRecord(kstart, 0);
  hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice,0);
  increment_kernel<<<blocks, threads,0,0>>>(d_a, value);
  hipMemcpyAsync(data, d_a, nbytes, hipMemcpyDeviceToHost,0);
  hipEventRecord(kend, 0);

  while (hipEventQuery(kend) == hipErrorNotReady)
  {
    counter++;
  }

  checkCudaErrors(hipFree(d_a));

  return counter;
}



TEST_CASE_METHOD(array_fixture, "simple_cuda_increment_works" ) {

  auto c = count_up(ints.data(), ints.size());
  
  REQUIRE(c != 0);
  REQUIRE(c > 10e6);
  
}
