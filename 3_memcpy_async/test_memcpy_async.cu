#include "hip/hip_runtime.h"
#include <chrono>

#include "catch.hpp"
#include "test_fixture.hpp"

#include "hip/hip_runtime_api.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

double increment_by_one(std::int32_t* data,
		      std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 8;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));
  
  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);

  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);
  increment_kernel<<<blocks, threads>>>(d_a, value);
  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end-start;

  checkCudaErrors(hipFree(d_a));

  return diff.count();
}

double streamed_increment_by_one(std::int32_t* data,
			       std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 8;
  const std::size chunk = size/2;
  const std::size chunk_bytes = chunk*sizeof(std::int32_t);
  
  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));
  
  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);

  int h_a* = nullptr;
  hipHostMalloc(h_a, nbytes);
  std::copy(data,
	    data+size,
	    h_a);
  
  auto start = std::chrono::high_resolution_clock::now();
  
  std::vector<hipStream_t> streams(2);
  for(hipStream_t &str : streams){
    hipStreamCreate(&str);
  }

  for(int i = 0;i<streams.size();++i){
    hipMemcpyAsync(d_a + i*chunk,
		    data+i*chunk,
		    chunk_bytes,
		    hipMemcpyHostToDevice,
		    streams[i]);
  }
  
  for(int i = 0;i<streams.size();++i){
    increment_kernel<<<blocks, threads,0,streams[i]>>>(d_a+i*chunk, value);
  }

  for(int i = 0;i<streams.size();++i){
    hipMemcpyAsync(data+i*chunk,
		    d_a + i*chunk,
		    chunk_bytes,
		    hipMemcpyDeviceToHost,
		    streams[i]);
  }

  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end-start;

  checkCudaErrors(hipFree(d_a));
  for(hipStream_t &str : streams){
    hipStreamDestroy(str);
  }

  return diff.count();
}

TEST_CASE_METHOD(array_fixture, "fixture_works" ) {
  REQUIRE(ints.size() != 0);
  REQUIRE(ints.empty() != true);

  REQUIRE(floats.size() != 0);
  REQUIRE(floats.empty() != true);
 
}

TEST_CASE_METHOD(array_fixture, "simple_cuda_increment_works" ) {

  increment_by_one(ints.data(), ints.size());
  
  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 8);
  
}

TEST_CASE_METHOD(array_fixture, "streamed_cuda_increment_works" ) {

  streamed_increment_by_one(ints.data(), ints.size());
  
  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 8);
  
}

TEST_CASE_METHOD(array_fixture, "omp_faster" ) {

  double ser_timing = increment_by_one(ints.data(), ints.size());
  double omp_timing = streamed_increment_by_one(ints.data(), ints.size());
  
  REQUIRE(omp_timing < ser_timing);
  
}

