#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "test_fixture.hpp"

#include "hip/hip_runtime_api.h"

#include "omp.h"

#include <iostream>
#include <chrono>

__global__ void increment_kernel(int *g_data, int inc_value)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

double increment_by_one(std::int32_t* data,
		      std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);


  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);
  increment_kernel<<<blocks, threads>>>(d_a, value);
  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);

  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}

double openmp_increment_by_one(std::int32_t* data,
                             std::size_t size)
{

  int nbytes = size * sizeof(std::int32_t);
  int value = 1;

  // allocate device memory
  int *d_a=0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks  = dim3(size / threads.x, 1);

  omp_set_num_threads(4);
  auto start = std::chrono::high_resolution_clock::now();

  hipMemcpy(d_a, data, nbytes, hipMemcpyHostToDevice);

#pragma omp parallel
  {
    std::cout << "increment_kernel " << omp_get_thread_num() << "/" << omp_get_num_threads() << "\n";
  increment_kernel<<<blocks, threads>>>(d_a, value);
  }

  hipMemcpy(data, d_a, nbytes, hipMemcpyDeviceToHost);
  auto end = std::chrono::high_resolution_clock::now();

  checkCudaErrors(hipFree(d_a));

  return (end - start).count();
}


TEST_CASE_METHOD(array_fixture, "simple_cuda_increment_works" ) {

  increment_by_one(ints.data(), ints.size());
  
  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 1);
  
}

TEST_CASE_METHOD(array_fixture, "omp_cuda_increment_works" ) {

  openmp_increment_by_one(ints.data(), ints.size());

  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 4);

}

TEST_CASE_METHOD(array_fixture, "compare_times" ) {

  auto serial = increment_by_one(ints.data(), ints.size());
  auto parallel = openmp_increment_by_one(ints.data(), ints.size());

  REQUIRE(ints[0] != 0);
  REQUIRE(ints[0] == 5);

  REQUIRE(parallel < serial);
}
